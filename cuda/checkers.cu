
#include <hip/hip_runtime.h>
#include<cstdio>
#include<vector>

struct checkers_point{
    int board[64];
    int how_much_children;
    checkers_point * children = NULL;
    checkers_point * next = NULL;
    checkers_point * prev = NULL;
    checkers_point * parent = NULL;
    bool min_max;
    int value;
    int player;
};

class Queue{
    private:
        checkers_point * first = NULL;
        checkers_point * last = NULL;
        int size = 0;
    public:
		__device__
			void add_one(checkers_point * point) {
				if(point == NULL)
					return;
				if(first == NULL) {
					this->first = point;
					this->last = point;
				}
				else {
					this->last->next = point;
					point->prev = this->last;
					this->last = point;
				}
				this->size = this->size + 1;
			}
        __device__
            void add(checkers_point * layer) {
				if(layer == NULL)
					return;
				int counter = 0;
                if(this->first == NULL) {
                    this->first = layer;
                }
                else {
                    this->last->next=layer;
                    layer->prev=this->last;
                }
				checkers_point * temp = layer;
				counter+=1;
				while(temp->next != NULL) {
					temp = temp->next;
					counter+=1;
				}
				this->last = temp;
				this->size = this->size + counter;
            }
		
		__device__
			checkers_point * pop() {
				checkers_point * firs,* seco;
				firs = this->first;
				if(firs == NULL)
					return NULL;
				else
					seco = firs->next;
				if(firs->parent != seco->parent) {
					firs->next = NULL;
					seco->prev = NULL;
				}
				this->first = seco;
				this->size = this->size - 1;
				return firs;
			}
		
		__device__
			bool empty() {
				return this->size == 0;
			}

		__device__
			void clean() {
				while(this->size > 0)
					this->pop();
			}
};


extern "C" {

__device__
    void ramification(checkers_point * ch2, int thid, int how_deep){
        int pseudo_rand = thid % 7 + 2;
        if (!(thid == 0 && how_deep == 1))
            printf("%d | %d | %d | %d\n", thid, ch2->value, pseudo_rand, ch2->parent->value);
        else {
            printf("%d | %d\n", thid, pseudo_rand);
        }
        ch2->how_much_children = pseudo_rand;
        ch2->children = new checkers_point;
        ch2->children->value = ch2->value*100+1;
        ch2->children->parent = ch2;
        ch2 = ch2->children;
        for (int j = 1; j < pseudo_rand; j++){
            ch2->next = new checkers_point;
            ch2->next->value = ch2->value+1;
            ch2->next->parent = ch2->parent;
            ch2->next->prev = ch2;
            ch2 = ch2->next;
        }
    }

__device__
    void delete_subtree(checkers_point * ch) {
        //detaching subtree from parent
        if(ch->parent != NULL) {
            if(ch->parent->children == ch) {
                ch->parent->children = ch->next;
                if(ch->next != NULL) {
                    ch->next->prev = NULL;
                }
            }   
            else {
                if(ch->prev != NULL) {
                    ch->prev->next = ch->next;
                }
                if(ch->next != NULL) {
                    ch->next->prev = ch->prev;
                }
            }
        } 
        
        //deleting all nodes in BFS order
        Queue Q;
        checkers_point * temp, * child;
        Q.add_one(ch);

        while(!Q.empty()) {
            temp = Q.pop();

            child = temp->children;
			if(child != NULL)
            	Q.add(child);
            delete temp;
         }
    }

__device__
    void change_tree_to_subtree(checkers_point * old_tree, checkers_point * new_tree, int thread_num) {
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        checkers_point ** V = new checkers_point *[thread_num];
        if(thid == 0){
            new_tree->parent = NULL;
            checkers_point * child = old_tree->children;
            checkers_point * temp;
            Queue Q;
            int count = 0;

            Q.add(child);
            while(!Q.empty() && count<thread_num) {
                temp = Q.pop();
                V[count] = temp;
                if(temp->children !=NULL)
                    Q.add(temp->children);
                count++;
            }
            Q.clean();

            temp = old_tree;
            old_tree = new_tree;
            delete temp;
        }
        __syncthreads();
        if(thid < thread_num) {
            checkers_point * my_child = V[thid];
            delete_subtree(my_child);
        }
    }

__global__
	void alpha_beta(checkers_point * ch, int thread_num){
	    if((blockIdx.x * blockDim.x)+threadIdx.x == 0)
	        printf("Wartosci: %d %d\n",ch->value,ch->children->value);
		change_tree_to_subtree(ch, ch->children, thread_num);
		if((blockIdx.x * blockDim.x)+threadIdx.x == 0)
		    printf("Wartosci: %d %d\n", ch->value, ch->children->value);
	}
    
__global__
    void create_tree(int n, checkers_point * ch, int how_deep){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        int find_me = thid;
        int count_group = n;
            __syncthreads();
        if (thid < n){
            checkers_point * ch2 = ch;
            for (int i = 0; i < how_deep; i++){
                if (find_me == 0 && i + 1 == how_deep){
                    ramification(ch2, thid, how_deep);
                }
                __syncthreads();
                if (i + 1 == how_deep)
                    break;
                count_group = count_group/ch2->how_much_children;
                int group = find_me/count_group;
                if (group >= ch2->how_much_children)
                    break;
                find_me = find_me % count_group;
                ch2 = ch2->children;
                for (int k = 0; k < group; k++)
                    ch2 = ch2->next;
                __syncthreads();
            }
        }
    }
    
    
__device__
    void print_tr(checkers_point * ch){
        if (ch == NULL)
            return;
        print_tr(ch->children);
        print_tr(ch->next);
        printf("%d\n", ch->value);
    }
        
__global__
    void print_tree(int n, checkers_point * ch, int i){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (thid == 0){
            printf("____\n");
            print_tr(ch);
            printf("____\n");
        }
    }

__global__
    void new_line(int n, checkers_point * ch, int i){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (thid == 0){
            printf("____\n");
        }
    }

}
