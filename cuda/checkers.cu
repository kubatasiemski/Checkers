
#include <hip/hip_runtime.h>
#include<cstdio>

#define EMPTY 0
#define WHITE 1
#define BLACK 2
#define queenW 11
#define queenB 22

struct checkers_point{
    int board[64];
    int how_much_children;
    checkers_point * children = NULL;
    checkers_point * next = NULL;
    checkers_point * prev = NULL;
    checkers_point * parent = NULL;
    bool min_max;
    int value;
    int player;
};

extern "C" {


__device__
int pawn_owner(int * tab, int x, int y){
    if (tab[x*8+y] == BLACK || tab[x*8+y] == queenB)
        return BLACK;
    if (tab[x*8+y] == WHITE || tab[x*8+y] == queenW)
        return WHITE;
    return EMPTY;
}

__device__
bool is_queen(int * tab, int x, int y){
	int n = 8;
    return (tab[x*n+y] == queenB || tab[x*n+y] == queenW);
}

__device__
bool is_a_pawn(int * tab, int x, int y){
    return !(tab[x*8+y] == EMPTY);
}

__device__
bool correct_kill(int * tab, int x, int y, int x1, int y1){
    if (!is_a_pawn(tab, x1, y1))
	return false;
    if (pawn_owner(tab, x, y) != pawn_owner(tab, x1, y1))
        return true;
    return false;
}


__device__
bool queen_way(int * tab, int x, int y, int x1, int y1){
    int own = pawn_owner(tab, x, y);
    int x_r = x > x1 ? -1 : 1, y_r = y > y1 ? -1 : 1;
    bool next_empty = false;
    x += x_r; y += y_r;
    while (x != x1){
        if (is_a_pawn(tab, x, y)){
            if (next_empty)
                return false;
            next_empty = true;
            if (pawn_owner(tab, x, y) == own)
                return false;
        } else {
            next_empty = false;
        }
        x += x_r; y += y_r;
    }
    return true;
}

__device__
bool is_move_correct(int * tab, int x, int y, int who, int x1, int y1){
    int n = 8;
    if (x < 0 || x >= n || x1 < 0 || x1 >= n || y < 0 || y >= n || y1 < 0 || y1 >= n ){
//	printf("WRONG VALUE");
        return false;
    }
    if (std::abs(x-x1) != std::abs(y-y1)){
//	printf("ABS PROBLEM");
        return false;
    }
    int pwn_wnr = pawn_owner(tab, x, y);
    if (pwn_wnr == EMPTY){
//	printf("PAWN OWNER EMPTY");
        return false;
    }
    if (pwn_wnr != who){
//	printf("pwn_wnr != who");
        return false;
    }
    if (is_a_pawn(tab, x1, y1)){
//	printf("pawn in _");
        return false;
    }
    if (x < x1 && who == WHITE && tab[x*n+y] != queenW){
//	printf("WHITE WRONG WAY");
        return false;
    }
    if (x > x1 && who == BLACK && tab[x*n+y] != queenB){
//	printf("BLACK WRONG WAY");
        return false;
    }
    if ((tab[x*n+y] == queenW || tab[x*n+y] == queenB) && (!queen_way(tab, x, y, x1, y1))){
//      printf("queen problem");
	return false;
    }
    if (!is_queen(tab, x, y) && std::abs((x-x1)) > 1 && !correct_kill(tab, x, y, (x1+x)/2, (y1+y)/2)){
//      printf("Correct kill problem");
	return false;
    }
    return true;
}

__device__
	void copy_board(checkers_point * ch, checkers_point * ch2){
		for (int i = 0; i < 64; i++){
			ch2->board[i] = ch->board[i];
		}
	}

__device__
	checkers_point * pawn(checkers_point * ch, int x, int y, int x1, int y1, bool &nxt, checkers_point * chprev, int & rand, bool iskillsomethingnow){
		if (chprev != NULL)
		return ch;
		int * tab = ch->board;
		if (ch->parent != NULL)
			tab = ch->parent->board;
		if (is_move_correct(tab, x, y, pawn_owner(tab, x, y), x1, y1) == true){
//			printf("correct ");
			checkers_point * chld, * chld_now;
                        if (!nxt){
//				printf("chld ");
                                ch->children = new checkers_point;
				ch->children->parent = ch;
				ch->children->prev = NULL;
				chld = ch->children;
                        } else {
//				printf("next ");
				ch->next = new checkers_point;
				ch->next->parent = ch->parent;
				ch->next->prev = ch;
                        	chld = ch->next;
			}
			chld->how_much_children = 0;
			chld->next = chld->children = NULL;
			if (chprev != NULL)
				copy_board(chprev, chld);
			else
				copy_board(chld->parent, chld);
			chld->parent->how_much_children++;
			chld->value = rand++;
                        chld->board[x1*8+y1] = chld->board[x*8+y];
                        chld->board[x*8+y] = EMPTY;
			if (iskillsomethingnow)
                        chld->board[(x+x1)/2*8+(y+y1)/2] = EMPTY;
			ch = chld;
			chld_now = ch;
			nxt = true;
//			printf("%d, %d -> %d, %d\n", x, y, x1, y1);
			/*
			if (iskillsomethingnow){
				if (ch->board[x1*8+y1] == WHITE){
		                	ch = pawn(ch, x1, y1, x1-2, y1-2, nxt, chld_now, rand, true);
        		        	ch = pawn(ch, x1, y1, x1-2, y1+2, nxt, chld_now, rand, true);
				} else {
		        	        ch = pawn(ch, x1, y1, x1+2, y1-2, nxt, chld_now, rand, true);
  		                	ch = pawn(ch, x1, y1, x1+2, y1+2, nxt, chld_now, rand, true);
				}
			}
			*/
		}
		return ch;
	}

__device__
    checkers_point * dismember_child(checkers_point * ch, int x, int y, int turn_no, bool &nxt, int &rand){
	checkers_point * chb = ch->parent;
	if (!nxt){
//		printf(" NO PARENT ");
		chb = ch;
	}
	/*
	printf("NR %d\n", chb->value);
	for (int i = 0; i < 64; i++)
		printf("%d ", chb->board[i]);
	printf("\n");
	*/
	switch(chb->board[x*8+y]){
	    case WHITE:
		if (turn_no % 2 == 0){
//		printf("WHITE ");
		ch = pawn(ch, x, y, x-1, y-1, nxt, NULL, rand, false);
                ch = pawn(ch, x, y, x-1, y+1, nxt, NULL, rand, false);
		ch = pawn(ch, x, y, x-2, y-2, nxt, NULL, rand, true);
		ch = pawn(ch, x, y, x-2, y+2, nxt, NULL, rand, true);
		}
		break;
	    case BLACK:
		if (turn_no % 2 == 1){
//		printf("BLACK %d %d", x, y);
		ch = pawn(ch, x, y, x+1, y-1, nxt, NULL, rand, false);
                ch = pawn(ch, x, y, x+1, y+1, nxt, NULL, rand, false);
		ch = pawn(ch, x, y, x+2, y-2, nxt, NULL, rand, true);
                ch = pawn(ch, x, y, x+2, y+2, nxt, NULL, rand, true);
		}
		break;
	    default:
		break;
	}
	return ch;
    }

__device__
//add global size
    void ramification(checkers_point * ch2, int thid, int how_deep){
	bool nxt = false;
	int rand = ch2->value;
	printf("!%d!\n", how_deep);
	for (int i = 0; i < 8*8; i++){
	    if (ch2->board[i] != EMPTY){
		ch2 = dismember_child(ch2, i/8, i % 8, how_deep, nxt, rand);
	    }
	}
    }

__global__
    void create_tree(int n, checkers_point * ch, int how_deep){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        int find_me = thid;
        int count_group = n;
            __syncthreads();
        if (thid < n){
            checkers_point * ch2 = ch;
            for (int i = 0; i < how_deep; i++){
                if (find_me == 0 && i + 1 == how_deep){
                    ramification(ch2, thid, how_deep);
                }
                __syncthreads();
                if (i + 1 == how_deep)
                    break;
                count_group = count_group/ch2->how_much_children;
                int group = find_me/count_group;
                if (group >= ch2->how_much_children)
                    break;
                find_me = find_me % count_group;
                ch2 = ch2->children;
                for (int k = 0; k < group; k++)
                    ch2 = ch2->next;
                __syncthreads();
            }
        }
    }

__device__
    void print_tr(checkers_point * ch){
        if (ch == NULL)
            return;
	if (ch->children != NULL){
	printf("(c) %d ", ch->value);
        print_tr(ch->children);
	}
	if (ch->next != NULL){
	printf("(n) %d ", ch->value);
        print_tr(ch->next);
	}
	if (ch->next == NULL && ch->children == NULL){
        printf("%d\n", ch->value);
	for (int i = 0; i < 64; i++){
		printf("%d", ch->board[i]);
		if (i % 8 == 7)
			printf("\n");
	}
	printf("\n");
	}
    }

__global__
    void print_tree(int n, checkers_point * ch, int i){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (thid == 0){
            printf("____\n");
            print_tr(ch);
            printf("____\n");
        }
    }

__global__
    void set_root(checkers_point * ch, int * tab, int size){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (thid == 0){
	    ch->value = 1;
	    ch->children = NULL;
	    ch->next = NULL;
	    ch->prev = NULL;
	    ch->how_much_children = 0;
	    for (int i = 0; i < size*size; ++i)
		ch->board[i] = tab[i]; 
        }
    }

__global__
    void copy_best_result(checkers_point * ch, int * tab, int size){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (thid == 0){
	//find the best board!
            for (int i = 0; i < 64; ++i)
                tab[i] = ch->board[i];
        }
    }

}
