
#include <hip/hip_runtime.h>
#include<cstdio>

#define EMPTY 0
#define WHITE 1
#define BLACK 2
#define queenW 11
#define queenB 22

struct checkers_point{
    int board[64];
    int how_much_children;
    checkers_point * children = NULL;
    checkers_point * next = NULL;
    checkers_point * parent = NULL;
    bool min_max;
    int value;
    int player;
};

extern "C" {

__device__
int pawn_owner(int * tab, int x, int y){
    if (tab[x*8+y] == BLACK || tab[x*8+y] == queenB)
        return BLACK;
    if (tab[x*8+y] == WHITE || tab[x*8+y] == queenW)
        return WHITE;
    return EMPTY;
}

__device__
bool is_queen(int * tab, int x, int y){
	int n = 8;
    return (tab[x*n+y] == queenB || tab[x*n+y] == queenW);
}

__device__
bool is_a_pawn(int * tab, int x, int y){
    return !(tab[x*8+y] == EMPTY);
}

__device__
bool correct_kill(int * tab, int x, int y, int x1, int y1){
    if (!is_a_pawn(tab, x1, y1))
	return false;
    if (pawn_owner(tab, x, y) != pawn_owner(tab, x1, y1))
        return true;
    return false;
}


__device__
bool queen_way(int * tab, int x, int y, int x1, int y1){
    int own = pawn_owner(tab, x, y);
    int x_r = x > x1 ? -1 : 1, y_r = y > y1 ? -1 : 1;
    bool next_empty = false;
    x += x_r; y += y_r;
    while (x != x1){
        if (is_a_pawn(tab, x, y)){
            if (next_empty)
                return false;
            next_empty = true;
            if (pawn_owner(tab, x, y) == own)
                return false;
        } else {
            next_empty = false;
        }
        x += x_r; y += y_r;
    }
    return true;
}

__device__
bool is_move_correct(int * tab, int x, int y, int who, int x1, int y1){
    int n = 8;
    if (x < 0 || x >= n || x1 < 0 || x1 >= n || y < 0 || y >= n || y1 < 0 || y1 >= n ){
	printf("WRONG VALUE");
        return false;
    }
    if (std::abs(x-x1) != std::abs(y-y1)){
	printf("ABS PROBLEM");
        return false;
    }
    int pwn_wnr = pawn_owner(tab, x, y);
    if (pwn_wnr == EMPTY){
	printf("PAWNOWNEREMPTY");
        return false;
    }
    if (pwn_wnr != who){
	printf("LOL");
        return false;
    }
    if (is_a_pawn(tab, x1, y1)){
	printf("pawn in _");
        return false;
    }
    if (x < x1 && who == WHITE && tab[x*n+y] != queenW){
	printf("WHITE WRONG WAY");
        return false;
    }
    if (x > x1 && who == BLACK && tab[x*n+y] != queenB){
	printf("BLACK WRONG WAY");
        return false;
    }
    if ((tab[x*n+y] == queenW || tab[x*n+y] == queenB) && (!queen_way(tab, x, y, x1, y1))){
        printf("queen problem");
	return false;
    }
    if (!is_queen(tab, x, y) && std::abs((x-x1)) > 1 && !correct_kill(tab, x, y, (x1+x)/2, (y1+y)/2)){
        printf("Correct kill problem");
	return false;
    }
    return true;
}

__device__
	void copy_board(checkers_point * ch, checkers_point * ch2){
		for (int i = 0; i < 64; i++){
			ch2->board[i] = ch->board[i];
		}
	}

__device__
	checkers_point * pawn(checkers_point * ch, int x, int y, int x1, int y1, bool &nxt, bool br){
		if (is_move_correct(ch->board, x, y, pawn_owner(ch->board, x, y), x1, y1) == true){
			printf("CORR!");
			checkers_point * chld;
                        if (!nxt){
				printf("NEXT");
                                ch->children = new checkers_point;
				ch->children->parent = ch;
				chld = ch->children;
                        } else {
				ch->next = new checkers_point;
				ch->next->parent = ch->parent;
                        	chld = ch->next;
			}
                        copy_board(chld->parent, chld);
			/*
			chld->value = 123;
                        chld->board[x1*8+y1] = chld->board[x*8+y];
                        chld->board[x*8+y] = EMPTY;
                        chld->board[(x+x1)/2*8+(y+y1)/2] = EMPTY;
			/*
			ch = chld;
			nxt = true;
			printf("%d, %d -> %d, %d\n", x, y, x1, y1);
			/*
			if (ch->board[x1*8+y1] == WHITE){
	                	ch = pawn(ch, x1, y1, x1-2, y1-2, nxt);
        	        	ch = pawn(ch, x1, y1, x1-2, y1+2, nxt);
			} else {
		                ch = pawn(ch, x1, y1, x1+2, y1-2, nxt);
  		                ch = pawn(ch, x1, y1, x1+2, y1+2, nxt);
			}
			*/
		}
		return ch;
	}

__device__
    checkers_point * dismember_child(checkers_point * ch, int x, int y, bool nxt, int turn_no){
	checkers_point * chb = ch->parent;
	if (chb == NULL){
		printf(" NO PARENT ");
		chb = ch;
	}
	printf("NR %d\n", chb->value); 
	switch(chb->board[x*8+y]){
	    case WHITE:
		if (turn_no % 2 == 0){
		printf("WHITE ");
		ch = pawn(ch, x, y, x-1, y-1, nxt, false);
                ch = pawn(ch, x, y, x-1, y+1, nxt, false);
		ch = pawn(ch, x, y, x-2, y-2, nxt, false);
		ch = pawn(ch, x, y, x-2, y+2, nxt, false);
		}
		break;
	    case BLACK:
		if (turn_no % 2 == 1){
		printf("BLACK %d %d", x, y);
		ch = pawn(ch, x, y, x+1, y-1, nxt, false);
  //              ch = pawn(ch, x, y, x+1, y+1, nxt, false);
//		ch = pawn(ch, x, y, x+2, y-2, nxt, false);
  //              ch = pawn(ch, x, y, x+2, y+2, nxt, false);
		}
		break;
	    default:
		break;
	}
	return ch;
    }

__device__
//add global size
    void ramification(checkers_point * ch2, int thid, int how_deep){
	bool nxt = false;
	printf("!%d!\n", how_deep);
	for (int i = 0; i < 8*8; i++){
	    if (ch2->board[i] != EMPTY){
		ch2 = dismember_child(ch2, i/8, i % 8, nxt, how_deep);
		nxt |= true;
	    }
	} 
	/*
        int pseudo_rand = thid % 7 + 2;
	
        if (!(thid == 0 && how_deep == 1))
            printf("%d | %d | %d | %d\n", thid, ch2->value, pseudo_rand, ch2->parent->value);
        else {
            printf("%d | %d\n", thid, pseudo_rand);
        }
	
        ch2->how_much_children = pseudo_rand;
        ch2->children = new checkers_point;
        ch2->children->value = ch2->value*100+1;
        ch2->children->parent = ch2;
        ch2 = ch2->children;
        for (int j = 1; j < pseudo_rand; j++){
            ch2->next = new checkers_point;
            ch2->next->value = ch2->value+1;
            ch2->next->parent = ch2->parent;
            ch2 = ch2->next;
        }
	*/
    }
    
__global__
    void create_tree(int n, checkers_point * ch, int how_deep){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        int find_me = thid;
        int count_group = n;
            __syncthreads();
        if (thid < n){
            checkers_point * ch2 = ch;
            for (int i = 0; i < how_deep; i++){
                if (find_me == 0 && i + 1 == how_deep){
                    ramification(ch2, thid, how_deep);
                }
                __syncthreads();
                if (i + 1 == how_deep)
                    break;
                count_group = count_group/ch2->how_much_children;
                int group = find_me/count_group;
                if (group >= ch2->how_much_children)
                    break;
                find_me = find_me % count_group;
                ch2 = ch2->children;
                for (int k = 0; k < group; k++)
                    ch2 = ch2->next;
                __syncthreads();
            }
        }
    }
    
    
__device__
    void print_tr(checkers_point * ch){
        if (ch == NULL)
            return;
        print_tr(ch->children);
        print_tr(ch->next);
        printf("%d\n", ch->value);
    }
        
__global__
    void print_tree(int n, checkers_point * ch, int i){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (thid == 0){
            printf("____\n");
            print_tr(ch);
            printf("____\n");
        }
    }

__global__
    void set_root(checkers_point * ch, int * tab, int size){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (thid == 0){
	    ch->value = 1;
//	    ch->children = NULL;
//	    ch->next = NULL;
	    for (int i = 0; i < size*size; ++i)
		ch->board[i] = tab[i]; 
        }
    }

__global__
    void copy_best_result(checkers_point * ch, int * tab, int size){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (thid == 0){
	//find the best board!
            for (int i = 0; i < 64; ++i)
                tab[i] = ch->board[i];
        }
    }

}
